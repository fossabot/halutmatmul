
#include <hip/hip_runtime.h>
extern "C" __global__ void halut_encode(const float X[],
                                        const float tree_info[],
                                        int group_ids[], int N, int D,
                                        int size) {
  // group_ids [N, C]
  // X [N, D]
  // tree_info [3 * B]
  //    [0:B-1]: dims
  //    [B:2B-1]: thresholds
  //    [2B:3B-1]: classes
  // size = N * C

  // grid_dim = (N // rows_per_block (+ 1), )
  // block_dim = (rows_per_block, C)

  // WILL BE CHANGED
  const int depth = 4;
  const int B = 16;
  const int K = 16;
  const int C = 32;
  // CHANGE END

  // B = leaf elements
  __shared__ float s_tree_info[C * 3 * B];

  // could be optimized be changing threadIdx.x, and threadIdx.y to optimize for
  // warp size
  const int cid = threadIdx.y;
  const int tree_info_offset = cid * 3 * B;
  const int row_offset_X = blockIdx.x * blockDim.x * D + threadIdx.x * D;
  const int tid = blockIdx.x * blockDim.x * blockDim.y +
                  threadIdx.x * blockDim.y + threadIdx.y;
  if (tid < size) {
    // load
    if (threadIdx.x == 0) {
      for (int i = 0; i < 3 * B; ++i) {
        s_tree_info[tree_info_offset + i] = tree_info[tree_info_offset + i];
      }
    }
    __syncthreads();
    int group_id = 0;

    float threshold = 0;
    float x = 0;
    int index_offset_helper = 1;
    for (int i = 0; i < depth; ++i) {
      int index_offset = index_offset_helper - 1;
      index_offset_helper *= 2;
      int dim = __float2int_rn(
          s_tree_info[tree_info_offset + index_offset + group_id]);
      threshold = s_tree_info[tree_info_offset + B + index_offset + group_id];
      x = X[row_offset_X + dim];
      group_id = group_id * 2 + (x > threshold ? 1 : 0);
    }
    int class_ =
        __float2int_rn(s_tree_info[tree_info_offset + 2 * B + group_id]);
    group_ids[tid] = class_ + cid * K;
  }
}